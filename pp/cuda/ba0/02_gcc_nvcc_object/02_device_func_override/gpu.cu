#include "hip/hip_runtime.h"
#include <stdio.h>
#include "data.h"

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

// __global__ = cpu    call, run on device
// __device__ = device call, run on device
// __host__   = cpu    call, run on    cpu
#define BLOCK_NUM 1
#define THREAD_NUM 4

__global__
void myFmadd(Data_t *db) 
{
    db[threadIdx.x].d = db[threadIdx.x].a * db[threadIdx.x].b + db[threadIdx.x].c;
}

void gpu(Data_t *db, int db_num) 
{
    Data_t *deviceWorkbuf;
    hipMalloc((void **)&deviceWorkbuf, db_num*sizeof(Data_t));

    hipMemcpy(deviceWorkbuf, db, db_num*sizeof(Data_t), hipMemcpyHostToDevice);
    myFmadd<<<BLOCK_NUM, db_num>>>(deviceWorkbuf);
    hipDeviceSynchronize();
    hipMemcpy(db, deviceWorkbuf, db_num*sizeof(Data_t), hipMemcpyDeviceToHost);

    hipFree(deviceWorkbuf);
}

class Data3 : public Data2 {
public:
    __device__ void run() {
        d = a * b + c;
    }
};

__global__
void myRun(Data3 *db) 
{
    db[threadIdx.x].run();
}

void gpu2(Data2 *db, int db_num) 
{
    Data2 *deviceWorkbuf;
    hipMalloc((void **)&deviceWorkbuf, db_num*sizeof(Data2));

    hipMemcpy(deviceWorkbuf, db, db_num*sizeof(Data2), hipMemcpyHostToDevice);
    myRun<<<BLOCK_NUM, db_num>>>(reinterpret_cast<Data3 *>(deviceWorkbuf));
    hipDeviceSynchronize();
    hipMemcpy(db, deviceWorkbuf, db_num*sizeof(Data2), hipMemcpyDeviceToHost);

    hipFree(deviceWorkbuf);
}