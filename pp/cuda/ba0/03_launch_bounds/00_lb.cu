
#include <hip/hip_runtime.h>
#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

// __global__ = cpu    call, run on device
// __device__ = device call, run on device
// __host__   = cpu    call, run on    cpu
#define BLOCK_NUM 1
#define THREAD_NUM 4


int main() {

    return 0;
}